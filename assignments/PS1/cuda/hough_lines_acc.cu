/**
 * Author: Kapil Gupta <kpgupta98@gmail.com>
 * Organization: XantheLabs
 * Created: January 2017
 */

#pragma once


#ifndef HOUGH_LINES_ACC_CU_
#define HOUGH_LINES_ACC_CU_

#define THETA_GRANULARITY		(4*100)
#define RHO_GRANULARITY     1

#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <vector>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../util.h"
#include "cuda_util.h"



bool hough_lines_acc(cv::Mat img_a_edges, std::vector<std::vector<int> > &hough_acc,\
   std::vector<double> &thetas, std::vector<double> &rhos, int rho_resolution, int theta_resolution) {





}






#endif  // HOUGH_LINES_ACC_CU_
